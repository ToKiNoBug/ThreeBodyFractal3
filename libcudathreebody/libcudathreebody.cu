#include "libcudathreebody.h"

#include <hip/hip_runtime.h>

void *allocate_device_memory(size_t bytes, int *errorcode = nullptr) noexcept {
  void *dptr = nullptr;
  hipError_t ce = hipMalloc(&dptr, bytes);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }

  return dptr;
}

void free_device_memory(void *device_ptr, int *errorcode = nullptr) noexcept {
  hipError_t ce = hipFree(&device_ptr);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
}

void memcpy_host_to_device(const void *host_ptr, void *device_ptr, size_t bytes,
                           int *errorcode = nullptr) noexcept {
  hipError_t ce =
      hipMemcpy(device_ptr, host_ptr, bytes, hipMemcpyHostToDevice);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
}

void memcpy_device_to_host(const void *device_ptr, void *host_ptr, size_t bytes,
                           int *errorcode = nullptr) noexcept {
  hipError_t ce =
      hipMemcpy(host_ptr, device_ptr, bytes, hipMemcpyDeviceToHost);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
}

void wait_for_device() noexcept { hipDeviceSynchronize(); }