#include "libcudathreebody.h"

#include <hip/hip_runtime.h>

#include "internal.h"

void *libcudathreebody::allocate_device_memory(size_t bytes,
                                               int *errorcode) noexcept {
  void *dptr = nullptr;
  hipError_t ce = hipMalloc(&dptr, bytes);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }

  return dptr;
}

bool libcudathreebody::free_device_memory(void *device_ptr,
                                          int *errorcode) noexcept {
  hipError_t ce = hipFree(&device_ptr);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}

bool libcudathreebody::memcpy_host_to_device(const void *host_ptr,
                                             void *device_ptr, size_t bytes,
                                             int *errorcode) noexcept {
  hipError_t ce =
      hipMemcpy(device_ptr, host_ptr, bytes, hipMemcpyHostToDevice);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}

bool libcudathreebody::memcpy_device_to_host(const void *device_ptr,
                                             void *host_ptr, size_t bytes,
                                             int *errorcode) noexcept {
  hipError_t ce =
      hipMemcpy(host_ptr, device_ptr, bytes, hipMemcpyDeviceToHost);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}

bool libcudathreebody::wait_for_device(int *errorcode) noexcept {
  hipError_t ce = hipDeviceSynchronize();
  if (errorcode != nullptr) {
    *errorcode = ce;
  }

  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}

bool libcudathreebody::run_cuda_simulations(
    const libthreebody::input_t *const inputs_host,
    libthreebody::result_t *const dest_host, void *buffer_input_device,
    void *buffer_result_device, size_t num, libthreebody::compute_options &opt,
    int *errorcode) {
  hipError_t ce;

  const int num_run_10 = 10 * ((num) / 10);

  printf("num_run_10 = %i\n", num_run_10);

  if (num_run_10 > 0) {
    ce = hipMemcpy(buffer_input_device, inputs_host,
                    sizeof(input_t) * num_run_10, hipMemcpyHostToDevice);
    if (ce != hipError_t::hipSuccess) {
      if (errorcode != nullptr) {
        *errorcode = ce;
      }
      return false;
    }

    libcudathreebody::simulate_10<<<num_run_10 / 10, 30>>>(
        (const input_t *)buffer_input_device, opt,
        (result_t *)buffer_result_device);
    printf("%i tasks added to gpu by %i blocks.\n", num_run_10,
           num_run_10 / 10);
  }

  for (int i = num_run_10; i < num; i++) {
    libthreebody::simulate_2(inputs_host[i], opt, &dest_host[i]);
  }

  if (num_run_10 > 0) {

    hipDeviceSynchronize();

    ce = hipMemcpy(dest_host, buffer_result_device,
                    sizeof(result_t) * num_run_10, hipMemcpyDeviceToHost);
    printf("GPU finished %i tasks.\n", num_run_10);
  }

  if (ce != hipError_t::hipSuccess) {
    if (errorcode != nullptr) {
      *errorcode = ce;
    }
    return false;
  }
  return true;
}