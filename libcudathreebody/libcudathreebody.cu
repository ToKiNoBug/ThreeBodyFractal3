#include "libcudathreebody.h"

#include <hip/hip_runtime.h>

bool libcudathreebody::is_device_ok(int *errorcode) noexcept {
  int num = 0;
  hipError_t ce = hipGetDeviceCount(&num);

  if (errorcode != nullptr) {
    *errorcode = ce;
  }

  if (num <= 0 || ce != hipError_t::hipSuccess) {
    return false;
  }

  return true;
}

void *libcudathreebody::allocate_device_memory(size_t bytes,
                                               int *errorcode) noexcept {
  void *dptr = nullptr;
  hipError_t ce = hipMalloc(&dptr, bytes);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }

  return dptr;
}

bool libcudathreebody::free_device_memory(void *device_ptr,
                                          int *errorcode) noexcept {
  hipError_t ce = hipFree(&device_ptr);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}

bool libcudathreebody::memcpy_host_to_device(const void *host_ptr,
                                             void *device_ptr, size_t bytes,
                                             int *errorcode) noexcept {
  hipError_t ce =
      hipMemcpy(device_ptr, host_ptr, bytes, hipMemcpyHostToDevice);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}

bool libcudathreebody::memcpy_device_to_host(const void *device_ptr,
                                             void *host_ptr, size_t bytes,
                                             int *errorcode) noexcept {
  hipError_t ce =
      hipMemcpy(host_ptr, device_ptr, bytes, hipMemcpyDeviceToHost);
  if (errorcode != nullptr) {
    *errorcode = ce;
  }
  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}

bool libcudathreebody::wait_for_device(int *errorcode) noexcept {
  hipError_t ce = hipDeviceSynchronize();
  if (errorcode != nullptr) {
    *errorcode = ce;
  }

  if (ce == hipError_t::hipSuccess) {
    return false;
  }
  return true;
}
